#include "hip/hip_runtime.h"
/* -------------------------------------------------------------------------- */
#include "simulation.hh"
#include "grid.hh"
/* -------------------------------------------------------------------------- */
#include <iostream>
#include <exception>
/* -------------------------------------------------------------------------- */

/* -------------------------------------------------------------------------- */
__global__ void compute_step_one_thread_per_row(Grid uo, Grid u, Grid f, float h)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int M = u.m();
    int N = u.n();
    
    if((i>0) && (i < M-1))
    {
        for(int j = 1; j<N-1; j++)
        {
            u(i, j) = 0.25 * (  uo(i - 1, j) + uo(i + 1, j) +
                                uo(i, j - 1) + uo(i, j + 1) - 
                                f(i, j) * h * h);
        }
    }
}

/* -------------------------------------------------------------------------- */
__global__ void compute_step_one_thread_per_column(Grid uo, Grid u, Grid f, float h)
{
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int M = u.m();
    int N = u.n();
    
    if((j>0) && (j < M-1))
    {
        for(int i = 1; i<M-1; i++)
        {
            u(i, j) = 0.25 * (  uo(i - 1, j) + uo(i + 1, j) +
                                uo(i, j - 1) + uo(i, j + 1) - 
                                f(i, j) * h * h);
        }
    }
}

/* -------------------------------------------------------------------------- */
__global__ void compute_step_one_thread_per_entry(Grid uo, Grid u, Grid f, float h)
{
    int j = blockIdx.x*blockDim.x + threadIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y;

    int M = u.m();
    int N = u.n();

    if((i>0) && (i < M-1) && (j>0) && (j < N-1))
    {
        u(i, j) = 0.25 * (  uo(i - 1, j) + uo(i + 1, j) +
                            uo(i, j - 1) + uo(i, j + 1) - 
                            f(i, j) * h * h);
    }
}

/* -------------------------------------------------------------------------- */
__global__ void compute_step_one_thread_per_entry_shared(Grid uo, Grid u, Grid f, float h)
{
    extern __shared__ float s[];
    int J = blockIdx.x*blockDim.x;
    int I = blockIdx.y*blockDim.y;

    int j = threadIdx.x;
    int i = threadIdx.y;

    int M = u.m();
    int N = u.n();

    int a,b;
    
    if((I+i>0) && (I+i < M-1) && (J+j>0) && (J+j < N-1))
    {
        for(int k = i*blockDim.x + j; k<(blockDim.x+2)*(blockDim.y+2); k+=blockDim.x*blockDim.y)
        {
            a = k/(blockDim.x+2);
            b = k%(blockDim.x+2);
            
            if((I-1 + a > 0) && (J-1 + b > 0) && (I-1 + a < M-1) && (J-1 + b < N-1))
            {
                s[k] = uo(I-1 + a, J-1 + b);
            }
        }

        __syncthreads();
        
        // Have to add a "+1" in both dimensions when using s since s[0] corresonds to uo(I-1, J-1)
        u(I+i, J+j) = 0.25 * (  s[(i - 1 + 1)*(blockDim.x+2) + j + 1] + s[(i + 1 + 1)*(blockDim.x+2) + j + 1] +
                                s[(i + 1)*(blockDim.x+2) + j - 1 + 1] + s[(i + 1)*(blockDim.x+2) + j + 1 + 1] - 
                                f(I+i, J+j) * h * h);
    }
}

/* -------------------------------------------------------------------------- */
void Simulation::compute_step(int block_size) {
    Grid & u = m_grids.current();
    Grid & uo = m_grids.old();

    int m = u.m();
    int n = u.n();

    if(m!=n)
    {
        std::cerr<<"Error, matrix must be square"<<std::endl;
        return;
    }

    double h = 1./n;

    // add the kernel call here

    
    // One thread per row

    int dimGrid = (n-1)/block_size + 1;
    int dimBlock = block_size;

    
    compute_step_one_thread_per_row<<<dimGrid, dimBlock>>>(uo, u, m_f, h);
    hipDeviceSynchronize();
    

    
    /*
    // One thread per column
    
    int dimGrid = (n-1)/block_size + 1;
    int dimBlock = block_size;
    

    compute_step_one_thread_per_column<<<dimGrid, dimBlock>>>(uo, u, m_f, h);
    hipDeviceSynchronize();
    */

    /*
    // One thread per entry
    
    dim3 dimGrid = dim3((n-1)/block_size + 1, (n-1)/block_size + 1);
    dim3 dimBlock = dim3(block_size, block_size);
    
    compute_step_one_thread_per_entry<<<dimGrid, dimBlock>>>(uo, u, m_f, h);
    hipDeviceSynchronize();
    */


    /*
    // One thread per entry shared memory
    
    dim3 dimGrid = dim3((n-1)/block_size + 1, (n-1)/block_size + 1);
    dim3 dimBlock = dim3(block_size, block_size);

    compute_step_one_thread_per_entry_shared<<<dimGrid, dimBlock, (block_size+2)*(block_size+2)*sizeof(float)>>>(uo, u, m_f, h);
    hipDeviceSynchronize();
    */

    auto error = hipGetLastError();
    if(error != hipSuccess) {
        throw std::runtime_error("Error Launching Kernel: "
                                 + std::string(hipGetErrorName(error)) + " - "
                                 + std::string(hipGetErrorString(error)));
    }
}
